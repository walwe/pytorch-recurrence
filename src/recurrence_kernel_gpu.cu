#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include <ATen/ATen.h>
namespace {
    /**
    * CUDA Kernel to calculate Recurrence Matrix for full batch
    */
    template <typename scalar_t>
    __global__ void recurrence_kernel(
        const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> in,
        torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> out,
        const int batchSize,
        const int size) {

        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int batch = blockIdx.z;

        if(i<size && j<size) {
          out[batch][0][j][i] = abs(in[batch][i]-in[batch][j]);
        }
    }
} // namespace

/**
* Calculate Recurrence Matrix for full batch
*/
void recurrence_matrix_calc(torch::Tensor in, torch::Tensor output, int batchSize, int size) {
    const int threads = 8;
    const dim3 threadsPerBlock(threads, threads);
    const int blockSize = ceil(size/float(threads));
    const dim3 numBlocks(blockSize, blockSize, batchSize);

    // Run kernel using tensor's data type
    AT_DISPATCH_ALL_TYPES(in.type(), "recurrence_kernel", ([&] {
       recurrence_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
            in.packed_accessor<scalar_t,2,torch::RestrictPtrTraits, size_t>(),
            output.packed_accessor<scalar_t,4,torch::RestrictPtrTraits, size_t>(),
            batchSize, size
       );
    }));

   auto err = hipGetLastError();
   if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
   }
}